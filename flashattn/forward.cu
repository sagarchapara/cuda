#include <stdio.h>
#include <stdlib.h>
#include <sys/resource.h>
#include <chrono>
#include <vector>
#include <string>
#include <fstream>
#include <iomanip>

#include <hip/hip_runtime.h>

#include <assert.h>
#include <hip/hip_math_constants.h>

__global__ void flash_attn_fwd(
    int B,
    int H,
    int N,
    int D,
    const float *Q,
    const float *K,
    const float *V,
    float *O,
    float scaling_factor,
    const bool *mask,
    float dropout_prob,
    const int M,
    int BC,
    int BR,
    float *l,
    float *m)
{

    int tc = (N + BC - 1) / BC;
    int tr = (N + BR - 1) / BR;

    extern __shared__ float smem[];
    float *Qi = smem;             // BR * D
    float *Ki = Qi + (BR * D);    // BC * D
    float *Vi = Ki + (BC * D);    // BC * D
    float *Mi = Vi + (BC * D);    // BR
    float *Li = Mi + BR;          // BR
    float *Oi = Li + BR;          // BR * D
    float *MiOld = Oi + (BR * D); // BR

    const int batch_offset = blockIdx.z * N * D;
    const int lane = threadIdx.x & 31;
    const int warp_id = threadIdx.x >> 5;
    const int num_warps = blockDim.x >> 5;

    const int i = blockIdx.x;

    if (i >= tr)
        return;

    // Load Qi and initialize Oi, Mi, Li
    for (int idx = threadIdx.x; idx < BR * D; idx += blockDim.x)
    {
        Qi[idx] = Q[batch_offset + i * BR * D + idx];
        Oi[idx] = 0.0f;
    }

    for (int r = threadIdx.x; r < BR; r += blockDim.x)
    {
        Mi[r] = -HIP_INF_F;
        Li[r] = 0.0f;
    }

    __syncthreads();

    for (int j = 0; j < tc; j++)
    {
        {
            // Load Kj, Vj
            const int base = batch_offset + j * BC * D;
            for (int idx = threadIdx.x; idx < BC * D; idx += blockDim.x)
            {
                Ki[idx] = K[base + idx];
                Vi[idx] = V[base + idx];
            }
        }

        __syncthreads();

        // Warp-per-row reduction: each warp scans all columns of its assigned rows in 32-wide tiles.
        // Eliminates the multi-stride shared-memory halving + barriers.
        // Assumptions:
        //  - blockDim.x is a multiple of 32.
        //  - BR can be >= num_warps; we iterate rows in a strided fashion per warp.
        //  - Scores are computed on-the-fly; we compute Mi[row] = max over columns.
        // lane/warp_id/num_warps already computed above

        for (int row = warp_id; row < BR; row += num_warps)
        {
            float vmax = -HIP_INF_F;
            // Iterate over column tiles of width 32.
            for (int base = 0; base < BC; base += 32)
            {
                int col = base + lane;

                if (col < BC)
                {
                    float sum = 0.0f;
                    for (int e = 0; e < D; ++e)
                    {
                        sum += Qi[row * D + e] * Ki[col * D + e];
                    }
                    vmax = fmaxf(vmax, sum * scaling_factor);
                }
            }
// In-warp reduce vmax.
#pragma unroll
            for (int offset = 16; offset > 0; offset >>= 1)
            {
                float other = __shfl_down_sync(0xffffffff, vmax, offset);
                vmax = fmaxf(vmax, other);
            }
            if (lane == 0)
            {
                MiOld[row] = Mi[row];
                Mi[row] = fmaxf(vmax, Mi[row]);
            }
        }

        __syncthreads();

        // Compute Li[row] = sum_j exp(score(row,j) - Mi[row])
        for (int row = warp_id; row < BR; row += num_warps)
        {
            float lsum = 0.0f;
            const float mval = Mi[row];
            const float mold = MiOld[row];
            const float scale = __expf(-mold + mval);

            for (int dbase = 0; dbase < D; dbase += 32)
            {
                int d = dbase + lane;
                if (d < D)
                    Oi[row * D + d] *= scale;
            }

            for (int base = 0; base < BC; base += 32)
            {
                int col = base + lane;

                float p = 0.0f;
                if (col < BC)
                {
                    float sum = 0.0f;
                    for (int e = 0; e < D; ++e)
                    {
                        sum += Qi[row * D + e] * Ki[col * D + e];
                    }
                    p = __expf(sum * scaling_factor - mval);
                    lsum += p;
                }

                for (int dbase = 0; dbase < D; dbase += 32)
                {
                    int d = dbase + lane;
                    if (d < D && col < BC)
                    {
                        Oi[row * D + d] += p * Vi[col * D + d];
                    }
                }
            }
// In-warp reduce lsum.
#pragma unroll
            for (int offset = 16; offset > 0; offset >>= 1)
            {
                float other = __shfl_down_sync(0xffffffff, lsum, offset);
                lsum += other;
            }
            if (lane == 0)
                Li[row] = Li[row] * __expf(MiOld[row] - Mi[row]) + lsum;
        }

        __syncthreads();
    }

    // write the Oi to global memory (warp-per-row)
    {
        for (int row = warp_id; row < BR; row += num_warps)
        {
            float Li_row = Li[row];
            for (int cbase = 0; cbase < D; cbase += 32)
            {
                int col = cbase + lane;
                if (col < D)
                {
                    int out_index = batch_offset + i * BR * D + row * D + col;
                    O[out_index] = Oi[row * D + col] / Li_row;
                }
            }
            if (lane == 0)
            {
                l[batch_offset / D + i * BR + row] = Li_row;
            }
        }
    }
}